#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

__global__ void MatAdd(float *A, float *B, float *C, int N) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < N) {
        C[index] = A[index] + B[index];
    }
}

int main() {

    //配列のサイズ
    int N = 256;
    size_t size = N * sizeof(float);

    //CPU側の配列確保
    float *A = (float*)malloc(size);
    float *B = (float*)malloc(size);
    float *C = (float*)malloc(size);

    for(int k=0; k < N; k++) {
        A[k] = 1;
        B[k] = 1;
    }

    //GPU側の配列
    float *g_A, *g_B, *g_C;
    hipMalloc((void**) &g_A, size);
    hipMalloc((void**) &g_B, size);
    hipMalloc((void**) &g_C, size);

    //CPUの配列をGPUにコピー
    hipMemcpy(g_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(g_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(g_C, C, size, hipMemcpyHostToDevice);


    //カーネル実行
    hipEvent_t start, stop;
    float e_time = 0.0;
    int dimBlock = 32;
    int dimGrid = (N + dimBlock - 1)/ dimBlock;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); // timer start
    MatAdd<<<dimGrid, dimBlock>>>(g_A, g_B, g_C, N);
    hipEventRecord(stop,0); // timer stop
    hipEventSynchronize(stop);
    hipEventElapsedTime(&e_time, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    // Read C from device memory
    hipMemcpy(C, g_C, size, hipMemcpyDeviceToHost);

    printf("\tElapsed time: %f [ms]\n",e_time);

    int flag = 0;

    for(int i=0; i < N; i++) {
        if(C[i] != 2.0) {
            
            flag = 1;
        }
    }

    if(flag == 0) {
        printf("answer is true.\n");
    } else {
        printf("answer is false.\n");
    }

    //GPUメモリの解放
    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C);

    //CPUメモリの解放
    free(A);
    free(B);
    free(C);

}