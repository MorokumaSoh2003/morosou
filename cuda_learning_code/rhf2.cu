#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>
//#include <memory>
#include <vector>
#include <algorithm>
#include <utility>
#include <string>
#include <list>
#include <iostream>
#include <iterator>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

//#include "cblas.h"
//#include "lapacke.h"

//#include "rhf.cuh"
#include "input.cuh"
#include "gpurhf.cuh"
#include "device.cuh"
#include "moc.cuh"

#define listsize 5

std::vector<short> countEachOrbitalNumber(PrimitiveShell* h_ps, 
                                          short nps, int8_t orbitals)
{
    std::vector<short> num_orbital(orbitals, 0);
    for (short i = 0; i < nps; ++i) {
        num_orbital[h_ps[i].orbital]++;
    }
    //printf("num_orbital[0]: %d\n", num_orbital[0]);
    //printf("num_orbital[1]: %d\n", num_orbital[1]);
    return num_orbital;
}

std::vector<int> countEachShellPairNumber(const std::vector<short>& num_orbital)
{
    const int8_t orbitals = num_orbital.size();
    short tid = 0;
    std::vector<int> num_eachpair(orbitals * (orbitals + 1) / 2, 0);
    for (int8_t La = 0; La < orbitals; ++La) {
        for (int8_t Lb = La; Lb < orbitals; ++Lb) {
            if (La == Lb) {
                num_eachpair[tid] = num_orbital[La] * (num_orbital[La] + 1) / 2;
            }
            else {
                num_eachpair[tid] = num_orbital[La] * num_orbital[Lb];
            }
            tid++;
        }
    }
    //printf("num_eachpair[0]: %d\n", num_eachpair[0]);
    return num_eachpair;
}


double* runMolecularOrbitalMethod(Atom* h_nuc, short nnuc, short nao, 
                                  PrimitiveShell* h_ps, short nps, 
                                  double* h_C, double cutoff, double df)
{
    for (int i = 0; i < nnuc; ++i) {
        h_nuc[i].coord.x /= ANGSTROM_PER_BOHR;
        h_nuc[i].coord.y /= ANGSTROM_PER_BOHR;
        h_nuc[i].coord.z /= ANGSTROM_PER_BOHR;
    }

    for (short i = 0; i < nps; ++i) {
        h_ps[i].R.x /= ANGSTROM_PER_BOHR;
        h_ps[i].R.y /= ANGSTROM_PER_BOHR;
        h_ps[i].R.z /= ANGSTROM_PER_BOHR;
    }

    // hack
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);
    hipsolverDnDestroy(cusolverH);

    int nelec = 0;
    for (short nid = 0; nid < nnuc; ++nid) {
        nelec += h_nuc[nid].charge;
    }
    const int sqnao = nao * nao;

    double* d_C;
    double* d_Eps;
    double* d_G;
    hipMalloc(&d_C, sizeof(double) * sqnao);
    hipMalloc(&d_Eps, sizeof(double) * sqnao);
    hipMalloc(&d_G, sizeof(double) * sqnao * sqnao);
    hipMemset(d_G, 0, sizeof(double) * sqnao * sqnao);
    hipMemcpy(d_C, h_C, sizeof(double) * sqnao, hipMemcpyHostToDevice);

    std::string scf = "stored";
    // Hartree-Fock method
    std::pair<double, float> result;
    result = rhf(h_nuc, nnuc, nao, nelec, h_ps, nps, df, cutoff, scf, d_C, d_Eps, d_G, h_C);



    /*
    // Integral transformation
    double* h_C;
    double* h_Eps;
    double* h_Gao;
    double* h_Gmo;
    hipHostMalloc(&h_C, sizeof(double) * sqnao);
    hipHostMalloc(&h_Eps, sizeof(double) * sqnao);
    hipHostMalloc(&h_Gao, sizeof(double) * sqnao * sqnao);
    hipHostMalloc(&h_Gmo, sizeof(double) * nao * nao * nao * nao);
    hipMemcpy(h_C, d_C, sizeof(double) * sqnao, hipMemcpyDeviceToHost);
    hipMemcpy(h_Eps, d_Eps, sizeof(double) * sqnao, hipMemcpyDeviceToHost);
    hipMemcpy(h_Gao, d_G, sizeof(double) * sqnao * sqnao, hipMemcpyDeviceToHost);
    //ao2mo(nao, h_C, h_Gao, h_Gmo);

     Møller–Plesset method
    double E_mp2 = 0.0;
    E_mp2 = mp2(nao, nelec, h_Gao, h_Eps, E_rhf);
    printf("E_mp2: %.12f\n", E_mp2);

    hipFree(h_C);
    hipFree(h_Eps);
    hipFree(h_Gao);
    hipFree(h_Gmo);
    /**/

    const double E_rhf = result.first;
    // device
    double* d_tmp;
    hipMalloc(&d_tmp, sizeof(double) * sqnao * sqnao);
    //atom2mol(nao, d_C, d_G, d_tmp);
    //atom2mol4mp2(nao, nelec, d_C, d_G, d_tmp);
    //atom2mol4mp2_grouped(nao, nelec, d_C, d_G, d_tmp);
    //atom2mol_dgemm(nao, d_C, d_G, d_tmp);

    std::chrono::system_clock::time_point start, stop;
    start = std::chrono::system_clock::now();
    ao2mo_dgemm(nao, d_C, d_G, d_tmp);
    stop = std::chrono::system_clock::now();
    auto dgemm_ms = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() / 1e3;
    printf("AO2MO (DGEMM): %.3f [ms]\n", dgemm_ms);

    //hipMemcpy(h_Gao, d_G, sizeof(double) * sqnao * sqnao, hipMemcpyDeviceToHost);

    // device MP2
    double E_mp2 = mp2d(nao, nelec, d_G, d_Eps, E_rhf);
    printf("E_mp2: %.12f\n", E_mp2);
    hipFree(d_tmp);



    hipFree(d_G);
    hipFree(d_C);
    hipFree(d_Eps);

    return nullptr;
}





std::pair<double, float> rhf(Atom* h_nuc, short nnuc, short nao, int nelec, 
                             PrimitiveShell* h_ps, short nps, 
                             double df, double cutoff, std::string scf, 
                             double* d_C, double* d_Eps, double* d_G, double* h_C)
{
    const int sqnao = nao * nao;

    // Step 2. calculate molecular integrals
    double* d_S;
    double* d_H;
    hipMalloc(&d_S, sizeof(double) * sqnao);
    hipMalloc(&d_H, sizeof(double) * sqnao);
    hipMemset(d_S, 0, sizeof(double) * sqnao);
    hipMemset(d_H, 0, sizeof(double) * sqnao);

    Atom* d_atom;
    PrimitiveShell* d_pshell;
    hipMalloc(&d_atom, sizeof(Atom) * nnuc);
    hipMalloc(&d_pshell, sizeof(PrimitiveShell) * nps);

    double* d_F;
    double* d_Fp;
    double* d_D;
    double* d_X;
    hipMalloc(&d_F, sizeof(double) * sqnao);
    hipMalloc(&d_Fp, sizeof(double) * sqnao);
    hipMalloc(&d_D, sizeof(double) * sqnao);
    hipMalloc(&d_X, sizeof(double) * sqnao);
    hipMemset(d_D, 0, sizeof(double) * sqnao);


    // Step 3.
    double* d_w_2d;
    hipMalloc(&d_w_2d, sizeof(double) * sqnao);
    hipMemset(d_w_2d, 0, sizeof(double) * sqnao);

    // generate the lookup table for the Boys function
    const size_t table_size = sizeof(double) * LUT_NUM_XI * (LUT_N_RANGE + LUT_K_MAX + 1);
    double* h_F_xi;
    double* d_F_xi;
    hipHostMalloc(&h_F_xi, table_size);
    hipMalloc(&d_F_xi, table_size);
    //hipMemset(h_F_xi, 0.0, table_size);
    generateTaylorTable(LUT_N_RANGE, LUT_K_MAX, LUT_XI_RANGE, LUT_XI_INTERVAL, h_F_xi);
    hipMemcpy(d_F_xi, h_F_xi, table_size, hipMemcpyHostToDevice);


    // hack
    //hipsolverHandle_t cusolverH = NULL;
    //hipsolverDnCreate(&cusolverH);
    //hipsolverDnDestroy(cusolverH);

    std::chrono::system_clock::time_point rhf_s, rhf_e;
    rhf_s = std::chrono::system_clock::now();

    //std::sort(CGTO, CGTO + nao, compareContractedGTO);
    std::sort(h_ps, h_ps + nps, comparePrimitiveShell);
    const int8_t orbitals = h_ps[nps - 1].orbital + 1;

    std::vector<short> num_orbital = \
        countEachOrbitalNumber(h_ps, nps, orbitals);
    std::vector<int> num_eachpair = \
        countEachShellPairNumber(num_orbital);

    // Step 1. calculate nuclear replusion energy
    double nuclearE = nuclearRepulsionEnergy(h_nuc, nnuc);

    hipMemcpy(d_atom, h_nuc, sizeof(Atom) * nnuc, hipMemcpyHostToDevice);
    hipMemcpy(d_pshell, h_ps, sizeof(PrimitiveShell) * nps, hipMemcpyHostToDevice);

    float onee = oneElectronIntegral(d_pshell, nps, d_S, d_H, d_atom, nnuc, nao, num_orbital, num_eachpair, d_F_xi);
    float twoe = twoElectronIntegral(h_ps, d_pshell, nps, d_G, nao, cutoff, num_orbital, num_eachpair, d_F_xi);
    //printf("1e integrals: %.3f [ms]\n", onee);

    deviceSymmetricDiagonalize(d_S, d_w_2d, nao);
    deviceTransformationMatrix(d_S, d_w_2d, d_X, nao);

    // Step 4. Initialize the density matrix P
    if (true) {
        deviceInitCoefficientMatrix(d_C, d_H, d_X, d_Eps, nao);
        //printf("initialized\n");
    }
    deviceBuildDensityMatrix(d_C, d_D, 0.0, nelec, nao);
    deviceBuildFockMatrix(d_F, d_G, d_D, d_H, nao);


    std::chrono::system_clock::time_point start, end;
    start = std::chrono::system_clock::now();
    double convergedE = selfConsistentField(d_S, d_H, d_X, d_F, d_Fp, d_C, d_D, 
                                            d_G, d_Eps, nao, 
                                            nelec, cutoff, scf);
    end = std::chrono::system_clock::now();
    auto scf_ms = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1e3;

    printf("2e integrals: %.3f [ms]\n", twoe);
    printf("SCF interations: %.3f [ms]\n", scf_ms);

    //printf("nuclearE: %.8f\n", nuclearE);
    double rhfE = convergedE + nuclearE;


    rhf_e = std::chrono::system_clock::now();
    auto rhf_ms = std::chrono::duration_cast<std::chrono::microseconds>(rhf_e - rhf_s).count() / 1e3;

    printf("Restricted Hartree Fock: %.3f [ms]\n", rhf_ms);
    printf("RHF energy: %.12f [hartree]\n", rhfE);

    hipFree(d_F);
    hipFree(d_D);
    hipFree(d_S);
    hipFree(d_H);
    hipFree(d_Fp);
    hipFree(d_X);
    hipFree(d_w_2d);
    hipFree(d_F_xi);
    hipFree(h_F_xi);
    hipFree(d_atom);
    hipFree(d_pshell);

    hipMemcpy(h_C, d_C, sizeof(double) * sqnao, hipMemcpyDeviceToHost);

    return std::make_pair(rhfE, rhf_ms);
}


double selfConsistentField(double* d_S, double* d_H, double* d_X, double* d_F, 
                           double* d_Fp, double* d_C, double* d_D, 
                           double* d_G, double* d_Eps, 
                           short nao, short nelec, 
                           double cutoff, std::string scf)
{
    short iteration = 0;
    double previousE = 0.0;
    double updatedE, deltaE;
    double* d_Cp = d_Fp;
    //double alpha = df;
    short max_iterations = 50;
    double* d_rhfE = nullptr;
    hipMalloc(&d_rhfE, sizeof(double));

    // double df;
    const int sqnao = nao * nao;
    // double* d_swap;
    double* d_F_old;
    double* d_D_old;
    hipMalloc(&d_F_old, sizeof(double) * sqnao);
    hipMalloc(&d_D_old, sizeof(double) * sqnao);
    hipMemset(d_F_old, 0, sizeof(double) * sqnao);
    hipMemset(d_D_old, 0, sizeof(double) * sqnao);

    double* d_Dt;
    double* d_Ft;
    double* d_tmp;
    hipMalloc(&d_Dt, sizeof(double) * nao * nao);
    hipMalloc(&d_Ft, sizeof(double) * nao * nao);
    hipMalloc(&d_tmp, sizeof(double) * nao * nao);

    double* d_e;
    hipMalloc(&d_e, sizeof(double) * nao * nao); 
    std::vector<double> h_e(nao * nao);
    std::vector<double> h_F(nao * nao);
    hipMemcpy(h_F.data(), d_F, sizeof(double) * nao * nao, hipMemcpyDeviceToHost);

    std::vector<std::vector<double>> h_F_list; 
    h_F_list.insert(h_F_list.begin(), h_F);

    std::vector<std::vector<double>> h_e_list; 

    //*
    while (iteration < max_iterations) {
        updatedE = deviceCalculateEnergy(d_D, d_H, d_F, d_rhfE, nao);
        //printf("i = %d: %.8f\n", iteration, updatedE);
        deltaE = std::abs(updatedE - previousE);
        if (deltaE < 1.0e-10) {
            break;
        }
        previousE = updatedE;

        deviceOrthogonalize(d_F, d_X, d_Fp, nao);
        deviceSymmetricDiagonalize(d_Fp, d_Eps, nao);
        deviceUpdateCoefficientMatrix(d_X, d_Cp, d_C, nao);

        // d_swap = d_D;
        // d_D = d_D_old;
        // d_D_old = d_swap;
        // d_swap = d_F;
        // d_F = d_F_old;
        // d_F_old = d_swap;
        //hipMemcpy(d_D_old, d_D, sizeof(double) * sqnao, hipMemcpyDeviceToDevice);
        //hipMemcpy(d_F_old, d_F, sizeof(double) * sqnao, hipMemcpyDeviceToDevice);
        deviceBuildDensityMatrix(d_C, d_D, 0, nelec, nao);
        deviceBuildFockMatrix(d_F, d_G, d_D, d_H, nao);
        // df = optimizeDampingFactor4RHF(d_D, d_D_old, d_F, d_F_old, d_Dt, d_Ft, d_tmp, nao);
        // // df = 1;
        // linearInterpolateMatrix(d_D, d_D_old, nao, df);
        // linearInterpolateMatrix(d_F, d_F_old, nao, df);
        // printf("iteration %d: %.12f (alpha = %lf)\n", iteration, updatedE, df);
        calError(d_S, d_D, d_F, d_e, nao);
        // hipMemcpy(h_e.data(), d_e, sizeof(double) * nao * nao, hipMemcpyDeviceToHost);
        // std::copy(h_e.begin(), h_e.end(), std::ostream_iterator<double>(std::cout, " "));
        // std::cout << std::endl;
        pushList(d_F, d_e, h_F_list, h_e_list, nao);

        // std::vector<double> d_e_1d(nao * nao);
        // for (const auto& row : h_e_list) h_e_1d.insert(h_e_1d.end(), row.begin(), row.end());
        
        double **d_e_list;
        hipMalloc((void**)&d_e_list, nao * nao * h_e_list.size() * sizeof(double));

        for (int i = 0; i < h_e_list.size(); i++)
        {
            hipMemcpy(d_e_list[i], h_e_list[i].data(), sizeof(double) * nao * nao, hipMemcpyHostToDevice);
        }
        hipFree(d_e_list);
        break;
        iteration++;
    }
    /**/
    hipFree(d_rhfE);
    hipFree(d_F_old);
    hipFree(d_D_old);
    hipFree(d_Dt);
    hipFree(d_Ft);
    hipFree(d_tmp);
    
    

    if (iteration == max_iterations) {
        printf("SCF procedure has not been converged...\n");
        return 0.0;
    }
    else {
        printf("Successfully converged (#iteration: %d)\n", iteration);
        return updatedE;
    }
}


void linearInterpolateMatrix(double* d_new, double* d_old, int nao, double df)
{
    double alpha = 1 - df;
    double beta = df;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);
    hipblasDgeam(cublasH, transa, transb, nao, nao,
                &alpha, d_old, nao, &beta, d_new, nao, d_new, nao);
    hipblasDestroy(cublasH);
}


double optimizeDampingFactor4RHF(double* d_Dn, double* d_Do, double* d_Fn, double* d_Fo, 
                                 double* d_Dt, double* d_Ft, double* d_tmp, short nao)
{
    double df, s, c;
    double* d_s;
    double* d_c;
    //double* d_Dt;
    //double* d_Ft;
    //double* d_tmp;
    hipMalloc(&d_s, sizeof(double));
    hipMalloc(&d_c, sizeof(double));
    //hipMalloc(&d_Dt, sizeof(double) * nao * nao);
    //hipMalloc(&d_Ft, sizeof(double) * nao * nao);
    //hipMalloc(&d_tmp, sizeof(double) * nao * nao);
    hipMemset(d_s, 0, sizeof(double));
    hipMemset(d_c, 0, sizeof(double));

    double alpha = 1;
    double beta = -1;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);
    // D_new - D_old
    hipblasDgeam(cublasH, transa, transb, nao, nao,
                &alpha, d_Dn, nao, &beta, d_Do, nao, d_Dt, nao);
    // F_new - F_old
    hipblasDgeam(cublasH, transa, transb, nao, nao,
                &alpha, d_Fn, nao, &beta, d_Fo, nao, d_Ft, nao);
    beta = 0;
    // F_old * (D_new - D_old)
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, 
                &alpha, d_Dt, nao, d_Fo, nao, &beta, d_tmp, nao);
    traceMatrix<<<1, nao>>>(d_tmp, d_s, nao);
    hipDeviceSynchronize();
    // (F_new - F_old) * (D_new - D_old)
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, 
                &alpha, d_Dt, nao, d_Ft, nao, &beta, d_tmp, nao);
    traceMatrix<<<1, nao>>>(d_tmp, d_c, nao);
    hipDeviceSynchronize();
    hipblasDestroy(cublasH);

    hipMemcpy(&s, d_s, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&c, d_c, sizeof(double), hipMemcpyDeviceToHost);
    if (c > -0.5 * s) {
        df = -0.5 * s / c;
    }
    else {
        df = 1;
    }

    hipFree(d_s);
    hipFree(d_c);
    //hipFree(d_Dt);
    //hipFree(d_Ft);
    //hipFree(d_tmp);

    return df;
}



void deviceUpdateCoefficientMatrix(double* d_X, double* d_Cp, double* d_C, 
                                   short nao) 
{
    //hipMemset(d_C, 0, sizeof(double) * nao * nao);
    const double alpha = 1.0;
    const double beta = 0.0;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, 
                &alpha, d_Cp, nao, d_X, nao, &beta, d_C, nao);
    hipblasDestroy(cublasH);
}



double euclideanDistance(const Coordinate& P, const Coordinate& Q) {
    double xd = P.x - Q.x;
    double yd = P.y - Q.y;
    double zd = P.z - Q.z;
    double distance = std::sqrt(xd * xd + yd * yd + zd * zd);
    return distance;
}



double nuclearRepulsionEnergy(Atom* atom, short nnuc)
{
    double rAB;
    double nuclear_energy = 0.0;
    for (short A = 0; A < nnuc; ++A) {
        for (short B = A + 1; B < nnuc; ++B) {
            rAB = euclideanDistance(atom[A].coord, atom[B].coord);
            nuclear_energy += atom[A].charge * atom[B].charge / rAB;
        }
    }
    return nuclear_energy;
}






void deviceSymmetricDiagonalize(double* d_A, double* d_w_2d, short nao)
{
    double* d_w_1d = nullptr;
    int* d_info = nullptr;
    int lwork = 0;
    double* d_work = nullptr;

    const int division = (nao + 32 - 1) / 32;
    dim3 blocks(division, division);
    dim3 threads(32, 32);
    matrixTransposeInPlace<<<blocks, threads>>>(d_A, nao);

    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);

    hipMalloc(&d_w_1d, sizeof(double) * nao);
    hipMalloc(&d_info, sizeof(int));
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
    hipsolverDnDsyevd_bufferSize(cusolverH, jobz, uplo, nao, d_A, 
                                nao, d_w_1d, &lwork);
    hipMalloc(&d_work, sizeof(double) * lwork);
    hipsolverDnDsyevd(cusolverH, jobz, uplo, nao, d_A, nao, 
                     d_w_1d, d_work, lwork, d_info);

    matrixTransposeInPlace<<<blocks, threads>>>(d_A, nao);

    short num_blocks = (nao + 256 - 1) / 256;
    deviceExpandEvalues1Dto2D<<<num_blocks , 256>>>(d_w_1d, d_w_2d, nao);

    hipFree(d_w_1d);
    hipFree(d_info);
    hipFree(d_work);
    hipsolverDnDestroy(cusolverH);
}


void deviceTransformationMatrix(double* d_U, double* d_w_2d, double* d_X, 
                                short nao)
{
    short num_blocks = (nao + 256 - 1) / 256;
    deviceDivideSquareRoot<<<num_blocks , 256>>>(d_w_2d, nao);   

    const double alpha = 1.0;
    const double beta = 0.0;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, 
                &alpha, d_w_2d, nao, d_U, nao, &beta, d_X, nao);
    
    hipblasDestroy(cublasH);
}


void deviceOrthogonalize(double* d_target, double* d_trans, double* d_Fp, 
                         short nao)
{
    const double alpha = 1.0;
    const double beta = 0.0;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);
    hipblasDgemm(cublasH, transa, HIPBLAS_OP_T, nao, nao, nao, 
                &alpha, d_target, nao, d_trans, nao, 
                &beta, d_Fp, nao);
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, 
                &alpha, d_trans, nao, d_Fp, nao, 
                &beta, d_Fp, nao);
    hipblasDestroy(cublasH);
}



void deviceInitCoefficientMatrix(double* d_C, double* d_H, double* d_X, 
                                 double* d_Eps, short nao)
{
    double* d_Hp;
    hipMalloc(&d_Hp, sizeof(double) * nao * nao);
    double* d_Cp = d_Hp;
    deviceOrthogonalize(d_H, d_X, d_Hp, nao);
    deviceSymmetricDiagonalize(d_Hp, d_Eps, nao);
    deviceUpdateCoefficientMatrix(d_X, d_Cp, d_C, nao);
    hipFree(d_Hp);
}


bool compareContractedGTO(const ContractedGTO& mu, const ContractedGTO& nu)
{
    short L_mu = mu.shell.l + mu.shell.m + mu.shell.n;
    short L_nu = nu.shell.l + nu.shell.m + nu.shell.n;
    if (L_mu != L_nu) {
        return L_mu < L_nu;
    }
    else {
        return mu.head < nu.head;
    }
}

bool comparePrimitiveShell(const PrimitiveShell& mu, const PrimitiveShell& nu)
{
    //*
    if (mu.orbital != nu.orbital) {
        return mu.orbital < nu.orbital;
    }
    else {
        return mu.cid < nu.cid;
    }
    /**/
    //return mu.orbital < nu.orbital;
}


void deviceBuildFockMatrix(double* d_F, double* d_G, 
                           double* d_D, double* d_H, 
                           short nao)
{
    const int8_t threadsPerWarp = 32;
    const int8_t warpsPerBlock = (nao + threadsPerWarp - 1) / threadsPerWarp;
    const short threadsPerBlock = threadsPerWarp * warpsPerBlock;
    if (threadsPerBlock > 1024) {
        printf("too many cgtos.\n");
        std::exit(EXIT_FAILURE);
    }
    const int num_blocks = nao * nao;
    //const int num_blocks = nao * (nao + 1) / 2;
    dim3 blocks(num_blocks);
    dim3 threads(threadsPerWarp, warpsPerBlock);
    squareFock<<<blocks, threads>>>(d_F, d_G, d_D, d_H, nao);

    /*
    hipblasHandle_t cublasH = NULL;
    const double alpha = 1.0;
    hipblasCreate(&cublasH);
    hipblasDaxpy(cublasH, num_blocks, &alpha, d_F, 1, d_H, 1);
    hipblasDestroy(cublasH);
    /**/
}


void deviceBuildDensityMatrix(double* d_C, double* d_D, double alpha, 
                              int nelec, short nao)
{
    const int sqnao = nao * nao;

    const int8_t threadsPerWarp = 32;
    const short threadsPerBlock = 256;
    const int8_t warpsPerBlock = threadsPerBlock / threadsPerWarp;
    const int num_blocks = (sqnao + threadsPerBlock - 1) / threadsPerBlock;
    dim3 blocks(num_blocks);
    dim3 threads(threadsPerWarp, warpsPerBlock);

    cudaDensityMatrix<<<blocks, threads>>>(d_C, d_D, alpha, nelec, nao);
}


double calculateRHFenergy(double* h_D, double* h_H, double* h_F, short nao)
{
    int cid;
    double sigma = 0.0;
    for (short mu = 0; mu < nao; ++mu) {
        for (short nu = 0; nu < nao; ++nu) {
            cid = nao * mu + nu;
            sigma += h_D[cid] * (h_H[cid] + h_F[cid]);
        }
    }
    return 0.5 * sigma;
}

//*
double deviceCalculateEnergy(double* d_D, double* d_H, double* d_F, 
                             double* d_rhfE, short nao) 
{
    const int sqnao = nao * nao;

    double rhfE = 0.0;
    hipMemcpy(d_rhfE, &rhfE, sizeof(double), hipMemcpyHostToDevice);

    const int8_t threadsPerWarp = 32;
    const short threadsPerBlock = 1024;
    const int8_t warpsPerBlock = threadsPerBlock / threadsPerWarp;
    const int num_blocks = (sqnao + threadsPerBlock - 1) / threadsPerBlock;
    dim3 blocks(num_blocks);
    dim3 threads(threadsPerWarp, warpsPerBlock);
    cudaCalculateEnergy<<<blocks, threads>>>(d_D, d_H, d_F, d_rhfE, nao);

    hipMemcpy(&rhfE, d_rhfE, sizeof(double), hipMemcpyDeviceToHost);

    //printf("%.8f\n", rhfE);

    return 0.5 * rhfE;
}
/**/

//DIIS
void calError(double* d_S, double* d_D, double* d_F, double* d_e, short nao)
{
    double* d_tmp;
    hipMalloc((void **)&d_tmp, sizeof(double) * nao * nao);
    
    const double alpha = 1.0;
    double beta = 0.0;
    hipblasHandle_t cublasH = NULL;
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasCreate(&cublasH);

    //F * P * S
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, &alpha, d_F, nao, d_D, nao, &beta, d_tmp, nao);
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, &alpha, d_tmp, nao, d_S, nao, &beta, d_e, nao);

    //S * P * F
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, &alpha, d_S, nao, d_D, nao, &beta, d_tmp, nao);
    beta = -1.0;
    hipblasDgemm(cublasH, transa, transb, nao, nao, nao, &alpha, d_tmp, nao, d_F, nao, &beta, d_e, nao);

}

void pushList(double* d_F, double* d_e, std::vector<std::vector<double>> h_F_list, std::vector<std::vector<double>> h_e_list, short nao)
{
    std::vector<double> h_F(nao * nao);
    std::vector<double> h_e(nao * nao);
    hipMemcpy(h_F.data(), d_F, sizeof(double) * nao * nao, hipMemcpyDeviceToHost);
    hipMemcpy(h_e.data(), d_e, sizeof(double) * nao * nao, hipMemcpyDeviceToHost);

    if(h_F_list.size() == listsize) h_F_list.pop_back();
    if(h_e_list.size() == listsize) h_e_list.pop_back();
    h_F_list.insert(h_F_list.begin(), h_F);
    h_e_list.insert(h_e_list.begin(), h_e);
}

// void calLambda(double** d_B, double* C, double* A, short size)
// {
//     int* d_pivot;
//     int* d_info;
//     double* d_work;
//     hipsolverHandle_t handle;
//     hipsolverDnCreate(&handle);
//     hipMalloc((void**)&d_pivot, size * sizeof(int));
//     hipMalloc((void**)&d_info, sizeof(int));

//     int workSize = 0;
//     hipsolverDnDgetrf_bufferSize(handle, size, size, d_B, size, &workSize);
//     hipMalloc((void**)&d_work, workSize * sizeof(double));

//     hipsolverDnDgetrf(handle, size, size, d_B, size, d_work, d_pivot, d_info);

//     hipsolverDnDgetrs(handle, HIPBLAS_OP_N, size, 1, d_B, size, d_pivot, A, size, d_info);
//     hipMemcpy(C, d_B, size * sizeof(double), hipMemcpyDeviceToHost);
// }

// __global__
// void newFock(double** d_F_list, double* d_F, double* C, short size, short num_CGTOs)
// {
//     const int tid = blockDim.x * threadIdx.y + threadIdx.x;
//     double ;
    
//     for (int i = 0; i < size; i++)
//     {
//        d_F[tid] += d_F_list[i][tid] * C[size];
//     }
// }