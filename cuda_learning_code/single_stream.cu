#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int *A, int *B, int *C) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < N) {
        int index1 = (index + 1) % 256;
        int index2 = (index + 2) % 256;
        float as = (A[index] + A[index1] + A[index2]) /3.0f;
        float bs = (B[index] + B[index1] + B[index2]) /3.0f;

        C[index] = (as + bs) / 2;

    }
}

int main() {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if(!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");

        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate( &start);
    hipEventCreate( &stop);

    hipStream_t stream;
    hipStreamCreate(&stream);

    int *A, *B, *C;
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    hipHostAlloc((void**)&A, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&B, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&C, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for(int i=0; i<FULL_DATA_SIZE; i++) {
        A[i] = 1;
        B[i] = 1;
    }

    hipEventRecord(start, 0);

    for(int i=0; i<FULL_DATA_SIZE; i+=N) {
        hipMemcpyAsync(dev_a, A+i, N*sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, B+i, N*sizeof(int), hipMemcpyHostToDevice, stream);

        kernel<<<N/256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

        hipMemcpyAsync(C+i, dev_c, N*sizeof(int), hipMemcpyDeviceToHost, stream);
    }

    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf("\tElapsed time: %f [ms]\n",elapsedTime);

    int flag = 0;

    for(int i=0; i < FULL_DATA_SIZE; i++) {
        if(C[i] != 1) {
            
            flag = 1;
        }
    }

    if(flag == 0) {
        printf("answer is true.\n");
    } else {
        printf("answer is false.\n");
    }

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipStreamDestroy(stream);
    
    return 0;
}