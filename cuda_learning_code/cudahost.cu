#include <stdio.h>
#include <hip/hip_runtime.h>

#define SIZE (64*1024*1024)

float cuda_host_alloc_test(int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapesedTime;

    hipEventCreate( &start);
    hipEventCreate( &stop);

    hipHostAlloc((void**)&a, size * sizeof(*a), hipHostMallocDefault);

    hipMalloc((void**)&dev_a, size * sizeof(*dev_a));

    hipEventRecord(start, 0);
    for(int i=0; i<100; i++) {
        if(up) {
            hipMemcpy(dev_a, a, sizeof(*a), hipMemcpyHostToDevice);
        }else {
            hipMemcpy(a, dev_a, sizeof(*a), hipMemcpyDeviceToHost);
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapesedTime, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf("\tElapsed time: %f [ms]\n",elapesedTime);

    hipHostFree(a);
    hipFree(dev_a);

    return elapesedTime;
}

int main() {
    float elapesedTime;
    float MB = (float)100*SIZE*sizeof(int)/1024/1024;

    elapesedTime = cuda_host_alloc_test(SIZE, true);
    printf("Time using cudaMalloc: %3.1f ms\n", elapesedTime);
    printf("\t MB/s during copy up: %3.1f\n", MB/(elapesedTime/1000));
}