#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int *A, int *B, int *C) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < N) {
        int index1 = (index + 1) % 256;
        int index2 = (index + 2) % 256;
        float as = (A[index] + A[index1] + A[index2]) /3.0f;
        float bs = (B[index] + B[index1] + B[index2]) /3.0f;

        C[index] = (as + bs) / 2;

    }
}

int main() {
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if(!prop.deviceOverlap) {
        printf("Device will not handle overlaps, so no speed up from streams\n");

        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate( &start);
    hipEventCreate( &stop);

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);


    int *A, *B, *C;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    hipMalloc((void**)&dev_a0, N * sizeof(int));
    hipMalloc((void**)&dev_b0, N * sizeof(int));
    hipMalloc((void**)&dev_c0, N * sizeof(int));

    hipMalloc((void**)&dev_a1, N * sizeof(int));
    hipMalloc((void**)&dev_b1, N * sizeof(int));
    hipMalloc((void**)&dev_c1, N * sizeof(int));

    hipHostAlloc((void**)&A, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&B, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&C, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for(int i=0; i<FULL_DATA_SIZE; i++) {
        A[i] = 1;
        B[i] = 1;
    }

    hipEventRecord(start, 0);

    for(int i=0; i<FULL_DATA_SIZE; i+=N*2) {
        hipMemcpyAsync(dev_a0, A+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, A+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);

        hipMemcpyAsync(dev_b0, B+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, B+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);

        kernel<<<N/256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N/256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

        hipMemcpyAsync(C+i, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(C+i+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    printf("\tElapsed time: %f [ms]\n",elapsedTime);

    int flag = 0;

    for(int i=0; i < FULL_DATA_SIZE; i++) {
        if(C[i] != 1) {
            
            flag = 1;
        }
    }

    if(flag == 0) {
        printf("answer is true.\n");
    } else {
        printf("answer is false.\n");
    }

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    
    return 0;
}