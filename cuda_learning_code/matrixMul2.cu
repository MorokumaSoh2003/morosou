#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matMul(float *A, float *B, float *C, int N) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    float Cvalue = 0;
    for (int i = 0; i < N; i++) {
        Cvalue += A[row * N + i] * B[i * N + col];
    }
    C[row * N + col] = Cvalue;
}

int main() {

    //配列のサイズ
    int N = 1024;
    size_t size = N * N * sizeof(float);

    //CPU側の配列確保
    float *A = (float*)malloc(size);
    float *B = (float*)malloc(size);
    float *C = (float*)malloc(size);

    for(int k=0; k < N*N; k++) {
        A[k] = 1;
        B[k] = 1;
    }

    //GPU側の配列
    float *g_A, *g_B, *g_C;
    hipMalloc((void**) &g_A, size);
    hipMalloc((void**) &g_B, size);
    hipMalloc((void**) &g_C, size);

    //CPUの配列をGPUにコピー
    hipMemcpy(g_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(g_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(g_C, C, size, hipMemcpyHostToDevice);

    //カーネル実行
    hipEvent_t start, stop;
    float e_time = 0.0;
    dim3 dimGrid(N/32, N/32);
    dim3 dimBlock(32, 32);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0); // timer start
    matMul<<<dimGrid, dimBlock>>>(g_A, g_B, g_C, N);
    hipEventRecord(stop,0); // timer stop
    hipEventSynchronize(stop);
    hipEventElapsedTime(&e_time, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );

    // Read C from device memory
    hipMemcpy(C, g_C, size, hipMemcpyDeviceToHost);

    printf("\tElapsed time: %f [ms]\n",e_time);

    int flag = 0;

    for(int i=0; i < N*N; i++) {
        if(C[i] != N) {
            flag = 1;
        }
    }

    if(flag == 0) {
        printf("answer is true.\n");
    } else {
        printf("answer is false.\n");
    }

    //GPUメモリの解放
    hipFree(g_A);
    hipFree(g_B);
    hipFree(g_C);

    //CPUメモリの解放
    free(A);
    free(B);
    free(C);
}